// originates from Ruetsch/Oster: Getting Started with CUDA
// more C++-style by Haase

#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>

__global__ void inc_gpu(float *const a, int N);

__global__ void inc_gpu(float *const a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        a[idx] = a[idx] + 1;
}

__global__ void sum_gpu(float *const a, float *const b, float *const c, int N);

__global__ void sum_gpu(float *const a, float *const b, float *const c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        c[idx] = a[idx]+b[idx];
}

using namespace std;

int main(void)
{
    int const N = 1400;
    int const nBytes = N * sizeof(float);

    int const blockSize = 64;
    int const numBlocks = (N + blockSize - 1) / blockSize;


    float *a_h = new float [nBytes];     // host data
    float *b_h = new float [nBytes];     // host data
    float *c_h = new float [nBytes];
    float *a_d, *b_d, *c_d;                    // device data
    hipMalloc((void **) &a_d, nBytes);
    hipMalloc((void **) &b_d, nBytes);
    hipMalloc((void **) &c_d, nBytes);

    for (int i = 0; i < N; i++)
        a_h[i] = 100.0f + static_cast<float>(i);

    hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);    //  a_d <- a_h
    hipMemcpy(b_d, a_d, nBytes, hipMemcpyDeviceToDevice);  //  b_d <- a_d
    hipMemcpy(c_d, a_d, nBytes, hipMemcpyDeviceToDevice);

// ---------------------------------------------------------

// Manipulate on GPU
    inc_gpu <<< numBlocks, blockSize>>>(b_d, N);             //  b_d := b_d+1.0    
    cout << hipGetErrorName(hipGetLastError()) << endl;

    sum_gpu <<< numBlocks, blockSize>>>(a_d,b_d,c_d,N);
    cout << hipGetErrorName(hipGetLastError()) << endl;

    hipMemcpy(b_h, b_d, nBytes, hipMemcpyDeviceToHost);    //  b_h <- b_d
    hipMemcpy(c_h, c_d, nBytes, hipMemcpyDeviceToHost);

// Check on CPU
    for (int i = 0; i < N; i++)
        assert( a_h[i] == b_h[i] - 1.0f );
    cout << "Check 1  OK" << endl;
    for (int i = 0; i < N; i++)
        assert(c_h[i] == a_h[i] + b_h[i]);
    cout << "Check 2  OK" << endl;
// ---------------------------------------------------------

    delete [] b_h;
    delete [] a_h;
    hipFree(a_d);
    hipFree(b_d);

    return 0;
}
