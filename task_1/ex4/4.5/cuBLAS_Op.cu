#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include "cuBLAS_Op.cuh"

cuBLAS_Op::cuBLAS_Op(hipblasHandle_t _c_handle) :
    c_handle(_c_handle)
{}

cuBLAS_Op::~cuBLAS_Op() { hipblasDestroy(this->c_handle); }

void    cuBLAS_Op::saxpy(const float a, cuBLAS_Vec &x, cuBLAS_Vec &y)
{
    _cublas_wrapper(hipblasSaxpy(this->c_handle, x.len, &a, x.v, 1, y.v, 1));
}

void    cuBLAS_Op::r_saxpy(const float a, cuBLAS_Vec &x, cuBLAS_Vec &y)
{
    this->saxpy(a, y, x);
}

void    cuBLAS_Op::axpby(const float a, const float b, cuBLAS_Vec &x,
        cuBLAS_Vec &y, cuBLAS_Vec &z)
{
    _cublas_wrapper(hipblasScopy(this->c_handle, x.len, y.v, 1, z.v, 1));
    _cublas_wrapper(hipblasSscal(this->c_handle, x.len, &b, z.v, 1));
    this->saxpy(1, x, z);
}

float   cuBLAS_Op::dot(cuBLAS_Vec &x, cuBLAS_Vec &y)
{
    float   ret;
    float   *tmp;

    hipMallocManaged(&tmp, sizeof(float));
    _cublas_wrapper(hipblasSdot(c_handle, x.len, x.v, 1, y.v, 1, tmp));
    ret = *tmp;
    hipFree(tmp);
    return ret;
}

float   cuBLAS_Op::norm(cuBLAS_Vec &x)
{
    float   ret;
    float   *tmp;

    hipMallocManaged(&tmp, sizeof(float));
    _cublas_wrapper(hipblasSnrm2(this->c_handle, x.len, x.v, 1, tmp));
    ret = *tmp;
    hipFree(tmp);
    return ret;
}

void    cuBLAS_Op::Mx(cuBLAS_Vec &M, cuBLAS_Vec &x, cuBLAS_Vec &r)
{
    _cublas_wrapper(hipblasSgemv(this->c_handle, HIPBLAS_OP_N, M.m, M.n,
                &this->_one, M.v, M.m, x.v, 1, &this->_zero, r.v, 1));
}

void    cuBLAS_Op::Mtx(cuBLAS_Vec &M, cuBLAS_Vec &x, cuBLAS_Vec &r)
{
    _cublas_wrapper(hipblasSgemv(this->c_handle, HIPBLAS_OP_T, M.m, M.n,
                &this->_one, M.v, M.m, x.v, 1, &this->_zero, r.v, 1));
}

void    cuBLAS_Op::tri_Mx(cuBLAS_Vec &M, cuBLAS_Vec &x, cuBLAS_Vec &r)
{
    _cublas_wrapper(hipblasSsbmv(this->c_handle, HIPBLAS_FILL_MODE_LOWER,
                M.n, 1, &this->_one, M.v, M.m, x.v, 1, &this->_zero, r.v, 1));
}

void    cuBLAS_Op::MMt(cuBLAS_Vec &M, cuBLAS_Vec &r)
{
    _cublas_wrapper(hipblasSgemm(this->c_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                M.m, M.n, M.n, &this->_one, M.v, M.m, M.v, M.m, &this->_zero, r.v, r.m));
}

void    cuBLAS_Op::Diag_M(cuBLAS_Vec &M, cuBLAS_Vec &r)
{
    _cublas_wrapper(hipblasScopy(this->c_handle, r.m, M.v, M.m+1, r.v, 1));
}

void    cuBLAS_Op::_cublas_wrapper(hipblasStatus_t status)
{
    try {
        if (status)
            throw std::exception();
    }
    catch (std::exception &e) {
        std::cerr << cublasGetStatusName(status) << ": " << cublasGetStatusString(status) << std::endl;
    }
    hipDeviceSynchronize();
}
