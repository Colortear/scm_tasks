// originates from Ruetsch/Oster: Getting Started with CUDA
// more C++-style by Haase

#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <random>
#include <cstdio>

__global__ void cmp_gpu(bool *ret, float *const a, float *const b, int N)
{
    extern __shared__ bool  sdata[];

    const int   idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int   str = gridDim.x * blockDim.x;
    bool        cmp = true;

    for (int i = idx; i < N; i += str)
        if (a[i] != b[i])
            cmp = false;
    sdata[threadIdx.x] = cmp;
    if (!cmp)
        printf("yes\n");
    __syncthreads();
    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {
        if (threadIdx.x <= s && sdata[threadIdx.x] && !sdata[threadIdx.x + s])
            sdata[threadIdx.x] = sdata[threadIdx.x + s];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        *ret = sdata[0];
}

using namespace std;

template <class T>
class CUDA_vec {
public:

    CUDA_vec() {};
    CUDA_vec(const int N, T *init)
    {
        hipMallocManaged(&this->v, N * sizeof(T));
        for (int i = 0; i < N; i++)
            this->v[i] = init[i];
    }
    ~CUDA_vec() { hipFree(v); }

    bool    cmp(const CUDA_vec &rhs, int const nb, int const bs, int const N)
    {
        bool    *result;
        bool    ret;

        hipMallocManaged(&result, sizeof(bool));
        cmp_gpu<<<nb, bs>>>(result, this->v, rhs.v, N);
        hipDeviceSynchronize();
        cout << hipGetErrorName(hipGetLastError()) << endl;
        ret = *result;
        hipFree(result);
        return ret;
    }

    T   *v;
};

int     main(void)
{
    int const       N = 1400;
    int const       blockSize = 64;
    int const       numBlocks = (N+blockSize-1)/blockSize;
    float           init_data[N];
    CUDA_vec<float> V[3];
    int             rand_idx;


    srand(time(0));
    for (int i = 0; i < N; i++)
        init_data[i] = static_cast<float>(rand() % 100);
    for (int i = 0; i < 3; i++)
        V[i] = CUDA_vec<float>(N, init_data);
    rand_idx = rand() % N;
    V[3].v[rand_idx] = V[1].v[rand_idx];
    assert(V[1].cmp(V[2], numBlocks, blockSize, N));
    cout << "test 1 is ok." << endl;
    assert(!V[1].cmp(V[3], numBlocks, blockSize, N));
    cout << "test 2 is ok.";
    return 0;
}
